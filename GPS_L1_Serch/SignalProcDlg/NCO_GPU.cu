#include "hip/hip_runtime.h"
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//  GPU��NCO���v�Z����B
//
// 
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdint.h>
#include "SP_Define.h "

#include "L1CaCode.h"

#include "L1Define.h "
//#include "GTS_global.h"
#include "GPU_L1_FreqPhase.h"
#include "L1Tracking.h"

#include "GPU_L1_FreqPhase.hpp"

//--#include "GPU_FIR.h"

#define NUM_THREADS		32      // Number of threads per block.
#define MAX_THREADS		32		// �������m�ۂŎg�p���� �ő�X���b�h��

// L1 CA Code
__constant__ int8_t devCaCode[1023];	// CA�R�[�h
uint16_t devCaCodeSv;					// CA�R�[�h �q���ԍ�

// RF�ݒ�
// �V�X�e���N���b�N
extern uint32_t gSysFreqRes;
// �L�����A���g��
extern uint32_t gL1CarrFreqRes;
// �R�[�h���g��(�n�[�t�`�b�v)
extern uint32_t gL1CodeFreqRes;

#if 0
// GP2010 �L�����A ���v���J
struct IQSignal IQ_Tble[8] = {
	 IQSignal(+1 ,+3) ,IQSignal(+3 ,+1) ,IQSignal(+3 ,-1) ,IQSignal(+1 ,-3)
	,IQSignal(-1 ,-3) ,IQSignal(-3 ,-1) ,IQSignal(-3 ,+1) ,IQSignal(-1 ,+3)};
#endif

__constant__ char i_table[] = {+1 ,+3 ,+3 ,+1 ,-1 ,-3 ,-3 ,-1};	// GP2010 �L�����A ���v���J
__constant__ char q_table[] = {+3 ,+1 ,-1 ,-3 ,-3 ,-1 ,+1 ,+3};

// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
// CA �T�[�`
//  x:�ʑ�
//  y:���g��
// 
// ThreadLen : 1�X���b�h����������T���v�����O��  
//				���ʑ����]�̐؂�ւ����܂����ꍇ������̂ŁA�g�p���ӁB
// 
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
__global__ void L1AccumLevel(float *AccumOut ,const uint32_t *SampleIn, int DataLen ,int width, GPU_L1_FreqPhase *Param)
{
	int si,posi;
	short inca;						// char���ƃ}�C�i�X�̊|���Z���ł��Ȃ��H�H
	uint16_t dump_co = 0;
	int16_t chip_val;
	int x = threadIdx.x + blockDim .x * blockIdx .x;
	int y =			      blockDim .y * blockIdx .y;

	int dout = x + width * y;		// ���X���b�h�̒S���ʒu
	int din = 0;					// ���̓f�[�^�擪�ʒu	- 0�̂肵�땔��	(���L�������Ɋ֘A���āAblockIdx.x�Ԋu�ł̕ϓ��ɂȂ�܂��B)

	uint32_t code_chip;
	uint32_t code_phase,code_phase_old;
	uint32_t carr_phase;
	uint32_t code_freq,carr_freq;

	GPU_L1_FreqPhase &tparam = Param[0];

	int i_sum = 0;
	int q_sum = 0;
	float acum_sum = 0.0f;

    __shared__	int8_t s_data_in[MAX_THREADS*4];

	tparam.getCodePhase(code_chip ,code_phase ,x);
	carr_phase = tparam.getCarrPhase(x);
	code_freq = tparam.getCodeFreq(y);
	carr_freq = tparam.getCarrFreq(y);

	code_phase_old = code_chip;
	chip_val = devCaCode[code_chip];

	if(y==1){
		y=1;
	}
	__syncthreads();

	posi = 0;
	// Tap���[�v
	while(posi < DataLen){

		si = posi % (MAX_THREADS*4);
		if(si == 0){
			((uint32_t*)s_data_in)[threadIdx.x]	= SampleIn[din + threadIdx.x];
			din		= din + blockDim.x;
		}
		__syncthreads();

		// *** �|���Z ***
		inca = ((int8_t*)s_data_in)[si];
		inca *= chip_val;
		i_sum += inca * i_table[carr_phase>>29];
		q_sum += inca * q_table[carr_phase>>29];

		// ���T���v����
		code_phase += code_freq;
		if(code_phase < code_phase_old)	{
			code_chip++;
			chip_val = devCaCode[code_chip];
			if(code_chip == 1023){
				code_chip = 0;
				dump_co++;
				acum_sum += (float)sqrt((float)((i_sum*i_sum)+(q_sum*q_sum)));
				i_sum = 0;
				q_sum = 0;
			}
		}
		__syncthreads();
		code_phase_old = code_phase;
		carr_phase += carr_freq;

		posi++;
	}

	acum_sum += (float)sqrt((float)((i_sum*i_sum)+(q_sum*q_sum)));

	AccumOut[dout] = acum_sum;
}

// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//  
//
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
__global__ void L1Ca_aggregate(uint32_t *AccumOut ,const int32_t *iAccumIn ,const int32_t *qAccumIn , int width)
{
	int x = threadIdx.x + blockDim .x * blockIdx .x;
	int y =			      blockDim .y * blockIdx .y;

	int dout = x + width * y;		// ���X���b�h�̒S���ʒu

	int i_sum;
	int q_sum;

    __shared__	int32_t s_data_i[MAX_THREADS+2];
    __shared__	int32_t s_data_q[MAX_THREADS+2];

#if 0
	// E,P,L �̐M�������v���� i�~q
	if(threadIdx.x == 0){
		if(x == 0){
			s_data_i[threadIdx.x] = iAccumIn[dout+1023-1];
			s_data_q[threadIdx.x] = qAccumIn[dout+1023-1];
		}else{
			s_data_i[threadIdx.x] = iAccumIn[dout-1];
			s_data_q[threadIdx.x] = qAccumIn[dout-1];
		}
	}
	__syncthreads();
	s_data_i[threadIdx.x+1]	= iAccumIn[dout];
	s_data_q[threadIdx.x+1]	= qAccumIn[dout];

	if(threadIdx.x == (MAX_THREADS-1)){
		s_data_i[threadIdx.x+2]	= iAccumIn[dout+1];
		s_data_q[threadIdx.x+2]	= qAccumIn[dout+1];
	}
	__syncthreads();

	i_sum	= s_data_i[threadIdx.x] + s_data_i[threadIdx.x+1] + s_data_i[threadIdx.x+2];
	q_sum	= s_data_q[threadIdx.x] + s_data_q[threadIdx.x+1] + s_data_q[threadIdx.x+2];
#else
	// P�݂̂� i�~q
	i_sum = iAccumIn[dout];
	q_sum = qAccumIn[dout];
#endif
	AccumOut[dout] = (uint32_t)sqrt((float)(i_sum*i_sum) + (float)(q_sum*q_sum));
}


// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
// �ő�l��T��
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
int32_t maxChipPos(const float *InData ,size_t ChipLen)
{
	float max = InData[0];
	size_t  pos = 0,i;

	for(i=1;i<ChipLen;i++){
		if(InData[i]>max){
			max = InData[i];
			pos = i;
		}
	}
	return pos;
}
int32_t maxChipPos(const int32_t *InData ,size_t ChipLen)
{
	int32_t max = InData[0];
	size_t  pos = 0,i;

	for(i=1;i<ChipLen;i++){
		if(InData[i]>max){
			max = InData[i];
			pos = i;
		}
	}
	return pos;
}
int32_t maxChipPos(const int32_t *InData ,size_t FreqN ,size_t ChipLen)
{
	int32_t max = maxChipPos(InData ,ChipLen);
	size_t  pos = 0,i;

	for(i=1;i<ChipLen;i++){
		if(InData[i]>max){
			max = InData[i];
			pos = i;
		}
	}
	return pos;
	

}
extern int cuda_iDivUp(int a, int b);
#if 0
//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
    return (a + (b - 1)) / b;
}
#endif

extern std::vector<L1CaCode>	gL1CaCode;
extern L1TrackingCh				gL1TrackingCh[32];

float OutDataf[MAX_THREADS*64*40];
extern int32_t OutData[MAX_THREADS*64*40];
extern int32_t OutiData[MAX_THREADS*64*40];
extern int32_t OutqData[MAX_THREADS*64*40];
uint32_t gDumpCo[MAX_THREADS*64*40];

#if 1
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//  ��M���x���̂݌���B
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
__host__ int GPU_L1Accumlevel(const uint8_t *InData ,size_t DataLen)
{
	int32_t ch;
	int32_t ack,tmp2,max;
	uint32_t *dev_in=0;
	float *dev_out=0;
	int32_t *dev_iout=0;
	int32_t *dev_qout=0;

    GPU_L1_FreqPhase *dev_caparam = 0;
	GPU_L1_FreqPhase caparam,caparambk;
    hipError_t cudaStatus;
	int32_t width	= 2048;
	int32_t height	= 40;
	GPU_L1_ThreadParam check;

	caparam.Sv = 1;

	memset(gL1TrackingCh ,0 ,sizeof(gL1TrackingCh));

#if 1
	caparam.fromCenterFreq(gSysFreqRes ,gL1CarrFreqRes ,500 ,20);
//--	caparam.fromCenterCaPhase(gSysFreqRes ,1023/2+1 ,0x00000000 ,0x80000000 ,0);
//--	caparam.fromCenterCarrPhase(gSysFreqRes ,0 ,500 ,40);
	caparam.fromBaseCodePhase(gSysFreqRes ,1023-10 ,0 ,0x80000000 ,0);
//--	check.CodeFreq = caparam. Base.CodeFreq + (caparam.Delta.CodeFreq * 20);
//--	check.CarrFreq = caparam.Base.CarrFreq + (caparam.Delta.CarrFreq * 20);// caparam.getCarrFreq(20);

	 ack = caparam.getCarrFreq(20);
	 tmp2 = caparam.getCodeFreq(20);

#else

	caparam.Base.CodeChip = 1023 - 10;
	caparam.Base.CodePhase = 0x00000000;		// 0
	caparam.Base.CodeFreq = clkFreq(SYS_FREQ_RES ,CODE_FREQ_RES/2);
	caparam.Base.CarrPhase = 0x00000000;		// 0
	caparam.Base.CarrFreq = clkFreq(SYS_FREQ_RES ,CARR_FREQ_RES );

	caparam.Delta.CodeChip = 0;
	caparam.Delta.CodePhase = 0x80000000;		// 0.5 chip
	caparam.Delta.CodeFreq = 0;
	caparam.Delta.CarrPhase = 0;
	caparam.Delta.CarrFreq = clkFreq(SYS_FREQ_RES ,500);
#endif

	// ���̓f�[�^ ������
 	cudaStatus = hipMalloc((void**)&dev_in, DataLen * sizeof(uint8_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// �o�̓f�[�^ ������
	cudaStatus = hipMalloc((void**)&dev_out, width*height*sizeof(float)+8);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// �p�����[�^ ������
	cudaStatus = hipMalloc((void**)&dev_caparam, sizeof(GPU_L1_FreqPhase));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// ���̓f�[�^���R�s�[
	hipMemcpy((void*)dev_in ,(void*)InData ,DataLen * sizeof(uint8_t) ,hipMemcpyHostToDevice);

	// �p�����[�^ �R�s�[
	hipMemcpy((void*)dev_caparam ,(void*)&caparam ,sizeof(GPU_L1_FreqPhase) ,hipMemcpyHostToDevice);

	dim3 block(NUM_THREADS, 1, 1);
    dim3 grid(cuda_iDivUp(width ,block.x) ,cuda_iDivUp(height ,block.y), 1);

	for(ch=0;ch<32;ch++){
		L1TrackingCh &tack = gL1TrackingCh[ch];

		// C/A�R�[�h���R�s�[
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devCaCode ),(const int8_t*)(&gL1CaCode[ch][0]) ,1023 * sizeof(int8_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpyToSymbol failed!");
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		L1AccumLevel<<<grid, block>>>(dev_out ,dev_in ,DataLen ,width ,dev_caparam);


		// ���֌��ʃR�s�[
		hipMemcpy((void*)OutDataf ,(void*)dev_out ,width*height*sizeof(float) ,hipMemcpyDeviceToHost);
		ack = maxChipPos(OutDataf ,2048*40);
		
		tack.CodePhase		= ack % 2048;
		tack.CarrDcoAcquire	= ack / 2048;	// caparam.getCarrFreq(ack / 2048);

		tack.Sv = ch;
		tack.CdLI = OutDataf[ack];

//		memcpy(gL1TrackingCh[ch].iFF ,&OutiData[ack-16] ,NUM_THREADS * sizeof(float));
//		memcpy(gL1TrackingCh[ch].qFF ,&OutqData[ack-16] ,NUM_THREADS * sizeof(float));

		// �p�����[�^ �R�s�[
//		hipMemcpy((void*)&caparambk ,(void*)dev_caparam ,sizeof(GPU_L1_FreqPhase) ,hipMemcpyDeviceToHost);
	}

Error:
	hipFree(dev_in);
	hipFree(dev_out);
    hipFree(dev_caparam);

	return 0;
}
#endif


// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
// CA ���� GP2010�Q��
//
//  x:���g���A�ʑ�
//  y:GPU_L1_FreqPhase[] 1�v�f�g�p
// 
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
__global__ void L1Ca_corr(int32_t *DumpI ,int32_t *DumpQ ,uint32_t *DumpCo ,int32_t *AccumI ,int32_t *AccumQ ,const uint32_t *SampleIn, size_t DataLen ,int width, GPU_L1_FreqPhase *Param)
{
	int si,posi;
	short inca;						// char���ƃ}�C�i�X�̊|���Z���ł��Ȃ��H�H
	uint16_t dump_co = 0;
	int16_t chip_val;
	int x = threadIdx.x + blockDim .x * blockIdx .x;
	int y =			      blockDim .y * blockIdx .y;

	int dout = x + width * y;		// ���X���b�h�̒S���ʒu
	int din = 0;					// ���̓f�[�^�擪�ʒu	- 0�̂肵�땔��	(���L�������Ɋ֘A���āAblockIdx.x�Ԋu�ł̕ϓ��ɂȂ�܂��B)

	uint32_t code_chip;
	uint32_t code_phase,code_phase_old;
	uint32_t carr_phase;
	uint32_t code_freq,carr_freq;

	GPU_L1_FreqPhase &tparam = Param[y];

	int i_sum = AccumI[dout];
	int q_sum = AccumQ[dout];

    __shared__	int8_t s_data_in[MAX_THREADS*4];

	tparam.getCodePhase(code_chip ,code_phase ,x);
	carr_phase = tparam.getCarrPhase(y);
	code_freq = tparam.getCodeFreq(y);
	carr_freq = tparam.getCarrFreq(y);

	code_phase_old = code_chip;
	chip_val = devCaCode[code_chip];

	__syncthreads();

	posi = 0;
	// Tap���[�v
	while(posi < DataLen){

		si = posi % (MAX_THREADS*4);
		if(si == 0){
			((uint32_t*)s_data_in)[threadIdx.x]	= SampleIn[din + threadIdx.x];
			din		= din + blockDim.x;
		}
		__syncthreads();

		// *** �|���Z ***
		inca = ((int8_t*)s_data_in)[si];
		inca *= chip_val;
		i_sum += inca * i_table[carr_phase>>29];
		q_sum += inca * q_table[carr_phase>>29];

		// ���T���v����
		code_phase += code_freq;
		if(code_phase < code_phase_old)	{
			code_chip++;
			chip_val = devCaCode[code_chip];
			if(code_chip == 1023){
				code_chip = 0;
				dump_co++;
//--				if((i_sum==0) && (q_sum==0)){
//--					i_sum = 0;
//--				}
				DumpI[dout] = i_sum;
				DumpQ[dout] = q_sum;
				i_sum = 0;
				q_sum = 0;
//						if(threadIdx.x == 15){}	// �ǔ�����
			}
		}
		__syncthreads();
		code_phase_old = code_phase;
		carr_phase += carr_freq;

		posi++;
	}

	DumpCo[dout] = dump_co;

	AccumI[dout] = i_sum;
	AccumQ[dout] = q_sum;

	if(threadIdx.x == 0){
		tparam.Base.CodeChip = code_chip;
		tparam.Base.CodePhase = code_phase;
		tparam.Base.CarrPhase = carr_phase;
	}
}

// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//  1ms�T�[�`
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
__host__ int GPU_L1Search_gpu2(float *line1 ,const uint8_t *InData ,size_t DataLen)
{
	int32_t ch,i;
	int32_t tmp,tmp2,max;
	uint32_t *dev_in=0;
	uint32_t *dev_out=0;
	int32_t *dev_iout=0;
	int32_t *dev_qout=0;
	uint32_t *dev_dumpco=0;
	int32_t *dev_iaccum=0;
	int32_t *dev_qaccum=0;

    GPU_L1_FreqPhase *dev_caparam = 0;
	GPU_L1_FreqPhase caparam[40],caparambk;
    hipError_t cudaStatus;
	int32_t width	= 2048;
	int32_t height	= 40;

//	caparam.Sv = 1;

	memset(gL1TrackingCh ,0 ,sizeof(gL1TrackingCh));

#if 0
	caparam.fromCenterFreq(SYS_FREQ_RES ,CARR_FREQ_RES ,500 ,20);
	caparam.fromBaseCodePhase(SYS_FREQ_RES ,1023-10 ,0 ,0x80000000 ,0);
#else
	for(i=0;i<40;i++){
		GPU_L1_FreqPhase &para = caparam[i];
		para.Base.CodeChip = 1023 - 10;
		para.Base.CodePhase= 0x00000000;		// 0
		para.Base.CodeFreq = clkFreq<uint32_t>(gSysFreqRes ,gL1CodeFreqRes/2+(((i-20)*500)/CC_SCALE));
///		para.Base.CodeFreq = clkFreq(SYS_FREQ_RES ,CODE_FREQ_RES/2);
		para.Base.CarrPhase= 0x00000000;		// 0
		para.Base.CarrFreq = clkFreq<uint32_t>(gSysFreqRes ,gL1CarrFreqRes+((i-20)*500));
///		para.Base.CarrFreq = clkFreq(SYS_FREQ_RES ,CARR_FREQ_RES);

		para.Delta.CodeChip	= 0;
		para.Delta.CodePhase= 0x80000000;		// 0.5 chip
		para.Delta.CodeFreq = 0;
		para.Delta.CarrPhase= 0;
		para.Delta.CarrFreq = 0;	// clkFreq(SYS_FREQ_RES ,500);
	}

///	DataLen = 5714+3;
#endif

	// ���̓f�[�^ ������
 	cudaStatus = hipMalloc((void**)&dev_in, DataLen * sizeof(uint8_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// �o�̓f�[�^ ������
	cudaStatus = hipMalloc((void**)&dev_out, width*height*sizeof(uint32_t));	// ****
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	// DUMP I ������
	cudaStatus = hipMalloc((void**)&dev_iout, width*height*sizeof(int32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	// DUMP Q ������
	cudaStatus = hipMalloc((void**)&dev_qout, width*height*sizeof(int32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	// DUMP �J�E���^ ������
	cudaStatus = hipMalloc((void**)&dev_dumpco, width*height*sizeof(int32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// ACCUM I ������
	cudaStatus = hipMalloc((void**)&dev_iaccum, width*height*sizeof(int32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	// ACCUM Q ������
	cudaStatus = hipMalloc((void**)&dev_qaccum, width*height*sizeof(int32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// �p�����[�^ ������
	cudaStatus = hipMalloc((void**)&dev_caparam, height*sizeof(GPU_L1_FreqPhase));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// ���̓f�[�^���R�s�[
	hipMemcpy((void*)dev_in ,(void*)InData ,DataLen * sizeof(uint8_t) ,hipMemcpyHostToDevice);

	// �p�����[�^ �R�s�[
	hipMemcpy((void*)dev_caparam ,(void*)&caparam ,height*sizeof(GPU_L1_FreqPhase) ,hipMemcpyHostToDevice);

	dim3 block(NUM_THREADS, 1, 1);
    dim3 grid(cuda_iDivUp(width ,block.x) ,cuda_iDivUp(height ,block.y), 1);
	
	for(ch=0;ch<32;ch++){
		max = 0;

		// C/A�R�[�h���R�s�[
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devCaCode ),(const int8_t*)(&gL1CaCode[ch][0]) ,1023 * sizeof(int8_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpyToSymbol failed!");
			goto Error;
		}

		hipMemset(dev_iaccum , 0 ,width*height*sizeof(int32_t));
		hipMemset(dev_qaccum , 0 ,width*height*sizeof(int32_t));

		// Launch a kernel on the GPU with one thread for each element.
		L1Ca_corr<<<grid, block>>>(dev_iout ,dev_qout ,dev_dumpco ,dev_iaccum ,dev_qaccum ,dev_in , DataLen ,width ,dev_caparam);

		L1Ca_aggregate<<<grid, block>>>(dev_out ,dev_iout ,dev_qout ,width);	// i,q������

		// ���֌��ʃR�s�[
		hipMemcpy((void*)OutiData ,(void*)dev_iout ,width*height*sizeof(int32_t) ,hipMemcpyDeviceToHost);
		hipMemcpy((void*)OutqData ,(void*)dev_qout ,width*height*sizeof(int32_t) ,hipMemcpyDeviceToHost);
		hipMemcpy((void*)gDumpCo ,dev_dumpco		,width*height*sizeof(int32_t) ,hipMemcpyDeviceToHost);
		hipMemcpy((void*)OutData ,(void*)dev_out	,width*height*sizeof(int32_t) ,hipMemcpyDeviceToHost);
		tmp = maxChipPos(OutData ,2048*40);
		if(max<OutData[tmp]){
			max = OutData[tmp];
		}

		gL1TrackingCh[ch].Sv = ch;
		gL1TrackingCh[ch].CdLI = max;

		memcpy(gL1TrackingCh[ch].iFF ,&OutiData[tmp-16] ,NUM_THREADS * sizeof(float));
		memcpy(gL1TrackingCh[ch].qFF ,&OutqData[tmp-16] ,NUM_THREADS * sizeof(float));

		// �p�����[�^ �R�s�[
		hipMemcpy((void*)&caparambk ,(void*)dev_caparam ,sizeof(GPU_L1_FreqPhase) ,hipMemcpyDeviceToHost);
	}

//--	for(i=0;i<2048;i++){
//--		line1[i] = OutData[i+tmp-1000];
//--	}

Error:
	hipFree(dev_in);
	hipFree(dev_out);
	hipFree(dev_iout);
	hipFree(dev_qout);
	hipFree(dev_dumpco);
	hipFree(dev_iaccum);
	hipFree(dev_qaccum);
    hipFree(dev_caparam);

	return 0;
}