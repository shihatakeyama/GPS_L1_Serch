#include "hip/hip_runtime.h"
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
// GL_.cu
//
// OpenGL���g�p����̂ɒǉ����郉�C�u����
// glew64.lib
// C:\ProgramData\NVIDIA Corporation\CUDA Samples\v10.0\common\lib\x64
// 
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *

#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

#include "GPU_helper.h"


// L1 �M������
__global__ void generateSpectrumKernel(float2 *h0,
                                       float2 *ht,
                                       unsigned int in_width,
                                       unsigned int out_width,
                                       unsigned int out_height,
                                       float t,
                                       float patchSize)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int in_index = y*in_width+x;
    unsigned int in_mindex = (out_height - y)*in_width + (out_width - x); // mirrored
    unsigned int out_index = y*out_width+x;

}

// update height map values based on output of FFT
__global__ void updateHeightmapKernel(float  *heightMap,
                                      float2 *ht,
                                      unsigned int width,
									  float thetaco)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

#if 0
    // cos(pi * (m1 + m2))
    float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;

    heightMap[i] = ht[i].x * sign_correction;
#endif

#if 0
	heightMap[i] = sin((x + y)*0.1f)*0.1f;
#endif
#if 1
	heightMap[i] = sin((float)x * 0.2f) * sin((float)y*0.2f + thetaco) * 5.2f;
#endif

}
// �������Z�o
extern "C"
void cudaUpdateHeightmapKernel(float  *d_heightMap,
                               float2 *d_ht,
                               unsigned int width,
                               unsigned int height,
                               bool autoTest)
{
	static float thetaco = 0.0f;

    dim3 block(16, 16, 1);
    dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);

	updateHeightmapKernel <<<grid, block >>>(d_heightMap, d_ht, width, thetaco);

	thetaco = thetaco + 0.06f;
	if (thetaco > (HIP_PI_F * 2)){
		thetaco -= (HIP_PI_F * 2);
	}
}

// �A�e���v�Z����B
// generate slope by partial differences in spatial domain
__global__ void calculateSlopeKernel(float2 *slopeOut, const float *h, unsigned int width, unsigned int height ,float Sensitivity)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

    float2 slope = make_float2(0.0f, 0.0f);

    if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1)){
		slope.x= (h[i + 1] - h[i - 1]) *Sensitivity;			// 0.002f;				//	/ h[i]/1024;	// /64;
		slope.y = (h[i + width] - h[i - width]) * Sensitivity;	//0.002f;	// /64;
    }

    slopeOut[i] = slope;
}

// �X�΂��Z�o
extern "C"
void cudaCalculateSlopeKernel(float *hptr, float2 *slopeOut,
                              unsigned int width, unsigned int height  ,float Sensitivity)
{
    dim3 block(8, 8, 1);
    dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);

    calculateSlopeKernel<<<grid2, block>>>(slopeOut, hptr, width, height ,Sensitivity);
}
